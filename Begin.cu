#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "Begin.cuh"
#include "Runner.cuh"
#include <chrono>



Begin::Begin(char *tag)
{
    this->tag = tag;
}

Begin::~Begin()
{
}

char *Begin::getTag()
{
    return this->tag;
}

void Begin::setData(int *valuesA, int *valuesB, int *dest, int arraySize, Op operation)
{
    this->valuesA = valuesA;
    this->valuesB = valuesB;
    this->dest = dest;
    this->operation = operation;
    this->arraySize = arraySize;
}

void Begin::setParams(int gridSize, int blockSize)
{
    this->gridSize = gridSize;
    this->blockSize = blockSize;
}

void Begin::beginLaunch()

{
    hipMalloc((void **)&g_vA, sizeof(int) * (this->arraySize));
    hipMalloc((void **)&g_vB, sizeof(int) * (this->arraySize));
    hipMalloc((void **)&g_d, sizeof(int) * (this->arraySize));

  

    
    hipMemcpy((void *)g_vA, (void *)valuesA, (sizeof(int) * this->arraySize), hipMemcpyHostToDevice);
    hipMemcpy((void *)g_vB, (void *)valuesB, (sizeof(int) * this->arraySize), hipMemcpyHostToDevice);
    hipMemcpy((void *)g_d, (void *)dest, (sizeof(int) * this->arraySize), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    hipLaunchByPtr<<<gridSize, blockSize>>>(static_cast<int>(operation), this->arraySize , g_vA, g_vB, g_d);
    hipDeviceSynchronize();
    auto stop = std::chrono::high_resolution_clock::now();

    auto dur = std::chrono::duration_cast<std::chrono::milliseconds>(stop-start);

    std::cout << "Operation INNER completed in " << dur.count() << " milliseconds\n";

    
    hipMemcpy((void *)dest, (void *)g_d, sizeof(int) * (this->arraySize), hipMemcpyDeviceToHost);

    hipFree(g_vA);
    hipFree(g_vB);
    hipFree(g_d);

    


}

__global__ void hipLaunchByPtr(int operation, int arraySize, int *g_vA, int *g_vB, int *g_d)
{
    
    Runner run;


    if (operation == 0)
    {
        run.processAdd(g_vA, g_vB, g_d, arraySize);
    }
    else if (operation == 1)
    {
        run.processSub(g_vA, g_vB, g_d, arraySize);
    }
    else if (operation == 2)
    {
        run.processMult(g_vA, g_vB, g_d, arraySize);
    }
    else if (operation == 3)
    {
        run.processDivi(g_vA, g_vB, g_d, arraySize);
    }
    
}
